
#include <hip/hip_runtime.h>
/*! \file VL_3D_cuda.cu
 *  \brief Definitions of the cuda 3D VL algorithm functions. */

#ifdef CUDA
#ifdef VL

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "../utils/gpu.hpp"
#include "../utils/hydro_utilities.h"
#include "../global/global.h"
#include "../global/global_cuda.h"
#include "../integrators/VL_3D_cuda.h"
#include "../hydro/hydro_cuda.h"
#include "../reconstruction/pcm_cuda.h"
#include "../reconstruction/plmp_cuda.h"
#include "../reconstruction/plmc_cuda.h"
#include "../reconstruction/ppmp_cuda.h"
#include "../reconstruction/ppmc_cuda.h"
#include "../riemann_solvers/exact_cuda.h"
#include "../riemann_solvers/roe_cuda.h"
#include "../riemann_solvers/hllc_cuda.h"
#include "../io/io.h"
#include "../riemann_solvers/hll_cuda.h"

__global__ void Update_Conserved_Variables_3D_half(Real *dev_conserved, Real *dev_conserved_half, Real *dev_F_x, Real *dev_F_y,  Real *dev_F_z, int nx, int ny, int nz, int n_ghost, Real dx, Real dy, Real dz, Real dt, Real gamma, int n_fields, Real density_floor);



void VL_Algorithm_3D_CUDA(Real *d_conserved, Real *d_grav_potential, int nx, int ny, int nz, int x_off, int y_off,
    int z_off, int n_ghost, Real dx, Real dy, Real dz, Real xbound,
    Real ybound, Real zbound, Real dt, int n_fields, Real density_floor,
    Real U_floor, Real *host_grav_potential )
{

  //Here, *dev_conserved contains the entire
  //set of conserved variables on the grid
  //concatenated into a 1-d array

  int n_cells = nx*ny*nz;
  int ngrid = (n_cells + TPB - 1) / TPB;

  // set values for GPU kernels
  // number of blocks per 1D grid
  dim3 dim1dGrid(ngrid, 1, 1);
  //  number of threads per 1D block
  dim3 dim1dBlock(TPB, 1, 1);

  //host_grav_potential is NULL if not using GRAVITY
  temp_potential = host_grav_potential;

  if ( !memory_allocated ){

    // allocate memory on the GPU
    //CudaSafeCall( cudaMalloc((void**)&dev_conserved, n_fields*n_cells*sizeof(Real)) );
    dev_conserved = d_conserved;
    CudaSafeCall( cudaMalloc((void**)&dev_conserved_half, n_fields*n_cells*sizeof(Real)) );
    CudaSafeCall( cudaMalloc((void**)&Q_Lx,  n_fields*n_cells*sizeof(Real)) );
    CudaSafeCall( cudaMalloc((void**)&Q_Rx,  n_fields*n_cells*sizeof(Real)) );
    CudaSafeCall( cudaMalloc((void**)&Q_Ly,  n_fields*n_cells*sizeof(Real)) );
    CudaSafeCall( cudaMalloc((void**)&Q_Ry,  n_fields*n_cells*sizeof(Real)) );
    CudaSafeCall( cudaMalloc((void**)&Q_Lz,  n_fields*n_cells*sizeof(Real)) );
    CudaSafeCall( cudaMalloc((void**)&Q_Rz,  n_fields*n_cells*sizeof(Real)) );
    CudaSafeCall( cudaMalloc((void**)&F_x,   n_fields*n_cells*sizeof(Real)) );
    CudaSafeCall( cudaMalloc((void**)&F_y,   n_fields*n_cells*sizeof(Real)) );
    CudaSafeCall( cudaMalloc((void**)&F_z,   n_fields*n_cells*sizeof(Real)) );

    #if defined( GRAVITY )
    // CudaSafeCall( cudaMalloc((void**)&dev_grav_potential, n_cells*sizeof(Real)) );
    dev_grav_potential = d_grav_potential;
    #else
    dev_grav_potential = NULL;
    #endif

    // If memory is single allocated: memory_allocated becomes true and successive timesteps won't allocate memory.
    // If the memory is not single allocated: memory_allocated remains Null and memory is allocated every timestep.
    memory_allocated = true;

  }

    #if defined( GRAVITY ) && !defined( GRAVITY_GPU )
    CudaSafeCall( cudaMemcpy(dev_grav_potential, temp_potential, n_cells*sizeof(Real), cudaMemcpyHostToDevice) );
    #endif


    // Step 1: Use PCM reconstruction to put primitive variables into interface arrays
    hipLaunchKernelGGL(PCM_Reconstruction_3D, dim1dGrid, dim1dBlock, 0, 0, dev_conserved, Q_Lx, Q_Rx, Q_Ly, Q_Ry, Q_Lz, Q_Rz, nx, ny, nz, n_ghost, gama, n_fields);
    CudaCheckError();


    // Step 2: Calculate first-order upwind fluxes
    #ifdef EXACT
    hipLaunchKernelGGL(Calculate_Exact_Fluxes_CUDA, dim1dGrid, dim1dBlock, 0, 0, Q_Lx, Q_Rx, F_x, nx, ny, nz, n_ghost, gama, 0, n_fields);
    hipLaunchKernelGGL(Calculate_Exact_Fluxes_CUDA, dim1dGrid, dim1dBlock, 0, 0, Q_Ly, Q_Ry, F_y, nx, ny, nz, n_ghost, gama, 1, n_fields);
    hipLaunchKernelGGL(Calculate_Exact_Fluxes_CUDA, dim1dGrid, dim1dBlock, 0, 0, Q_Lz, Q_Rz, F_z, nx, ny, nz, n_ghost, gama, 2, n_fields);
    #endif //EXACT
    #ifdef ROE
    hipLaunchKernelGGL(Calculate_Roe_Fluxes_CUDA, dim1dGrid, dim1dBlock, 0, 0, Q_Lx, Q_Rx, F_x, nx, ny, nz, n_ghost, gama, 0, n_fields);
    hipLaunchKernelGGL(Calculate_Roe_Fluxes_CUDA, dim1dGrid, dim1dBlock, 0, 0, Q_Ly, Q_Ry, F_y, nx, ny, nz, n_ghost, gama, 1, n_fields);
    hipLaunchKernelGGL(Calculate_Roe_Fluxes_CUDA, dim1dGrid, dim1dBlock, 0, 0, Q_Lz, Q_Rz, F_z, nx, ny, nz, n_ghost, gama, 2, n_fields);
    #endif //ROE
    #ifdef HLLC
    hipLaunchKernelGGL(Calculate_HLLC_Fluxes_CUDA, dim1dGrid, dim1dBlock, 0, 0, Q_Lx, Q_Rx, F_x, nx, ny, nz, n_ghost, gama, 0, n_fields);
    hipLaunchKernelGGL(Calculate_HLLC_Fluxes_CUDA, dim1dGrid, dim1dBlock, 0, 0, Q_Ly, Q_Ry, F_y, nx, ny, nz, n_ghost, gama, 1, n_fields);
    hipLaunchKernelGGL(Calculate_HLLC_Fluxes_CUDA, dim1dGrid, dim1dBlock, 0, 0, Q_Lz, Q_Rz, F_z, nx, ny, nz, n_ghost, gama, 2, n_fields);
    #endif //HLLC
    #ifdef HLL
    hipLaunchKernelGGL(Calculate_HLL_Fluxes_CUDA, dim1dGrid, dim1dBlock, 0, 0, Q_Lx, Q_Rx, F_x, nx, ny, nz, n_ghost, gama, 0, n_fields);
    hipLaunchKernelGGL(Calculate_HLL_Fluxes_CUDA, dim1dGrid, dim1dBlock, 0, 0, Q_Ly, Q_Ry, F_y, nx, ny, nz, n_ghost, gama, 1, n_fields);
    hipLaunchKernelGGL(Calculate_HLL_Fluxes_CUDA, dim1dGrid, dim1dBlock, 0, 0, Q_Lz, Q_Rz, F_z, nx, ny, nz, n_ghost, gama, 2, n_fields);
    #endif //HLL
    CudaCheckError();


    // Step 3: Update the conserved variables half a timestep
    hipLaunchKernelGGL(Update_Conserved_Variables_3D_half, dim1dGrid, dim1dBlock, 0, 0, dev_conserved, dev_conserved_half, F_x, F_y, F_z, nx, ny, nz, n_ghost, dx, dy, dz, 0.5*dt, gama, n_fields, density_floor );
    CudaCheckError();


    // Step 4: Construct left and right interface values using updated conserved variables
    #ifdef PCM
    hipLaunchKernelGGL(PCM_Reconstruction_3D, dim1dGrid, dim1dBlock, 0, 0, dev_conserved_half, Q_Lx, Q_Rx, Q_Ly, Q_Ry, Q_Lz, Q_Rz, nx, ny, nz, n_ghost, gama, n_fields);
    #endif
    #ifdef PLMP
    hipLaunchKernelGGL(PLMP_cuda, dim1dGrid, dim1dBlock, 0, 0, dev_conserved_half, Q_Lx, Q_Rx, nx, ny, nz, n_ghost, dx, dt, gama, 0, n_fields);
    hipLaunchKernelGGL(PLMP_cuda, dim1dGrid, dim1dBlock, 0, 0, dev_conserved_half, Q_Ly, Q_Ry, nx, ny, nz, n_ghost, dy, dt, gama, 1, n_fields);
    hipLaunchKernelGGL(PLMP_cuda, dim1dGrid, dim1dBlock, 0, 0, dev_conserved_half, Q_Lz, Q_Rz, nx, ny, nz, n_ghost, dz, dt, gama, 2, n_fields);
    #endif //PLMP
    #ifdef PLMC
    hipLaunchKernelGGL(PLMC_cuda, dim1dGrid, dim1dBlock, 0, 0, dev_conserved_half, Q_Lx, Q_Rx, nx, ny, nz, n_ghost, dx, dt, gama, 0, n_fields);
    hipLaunchKernelGGL(PLMC_cuda, dim1dGrid, dim1dBlock, 0, 0, dev_conserved_half, Q_Ly, Q_Ry, nx, ny, nz, n_ghost, dy, dt, gama, 1, n_fields);
    hipLaunchKernelGGL(PLMC_cuda, dim1dGrid, dim1dBlock, 0, 0, dev_conserved_half, Q_Lz, Q_Rz, nx, ny, nz, n_ghost, dz, dt, gama, 2, n_fields);
    #endif
    #ifdef PPMP
    hipLaunchKernelGGL(PPMP_cuda, dim1dGrid, dim1dBlock, 0, 0, dev_conserved_half, Q_Lx, Q_Rx, nx, ny, nz, n_ghost, dx, dt, gama, 0, n_fields);
    hipLaunchKernelGGL(PPMP_cuda, dim1dGrid, dim1dBlock, 0, 0, dev_conserved_half, Q_Ly, Q_Ry, nx, ny, nz, n_ghost, dy, dt, gama, 1, n_fields);
    hipLaunchKernelGGL(PPMP_cuda, dim1dGrid, dim1dBlock, 0, 0, dev_conserved_half, Q_Lz, Q_Rz, nx, ny, nz, n_ghost, dz, dt, gama, 2, n_fields);
    #endif //PPMP
    #ifdef PPMC
    hipLaunchKernelGGL(PPMC_cuda, dim1dGrid, dim1dBlock, 0, 0, dev_conserved_half, Q_Lx, Q_Rx, nx, ny, nz, n_ghost, dx, dt, gama, 0, n_fields);
    hipLaunchKernelGGL(PPMC_cuda, dim1dGrid, dim1dBlock, 0, 0, dev_conserved_half, Q_Ly, Q_Ry, nx, ny, nz, n_ghost, dy, dt, gama, 1, n_fields);
    hipLaunchKernelGGL(PPMC_cuda, dim1dGrid, dim1dBlock, 0, 0, dev_conserved_half, Q_Lz, Q_Rz, nx, ny, nz, n_ghost, dz, dt, gama, 2, n_fields);
    #endif //PPMC
    CudaCheckError();


    // Step 5: Calculate the fluxes again
    #ifdef EXACT
    hipLaunchKernelGGL(Calculate_Exact_Fluxes_CUDA, dim1dGrid, dim1dBlock, 0, 0, Q_Lx, Q_Rx, F_x, nx, ny, nz, n_ghost, gama, 0, n_fields);
    hipLaunchKernelGGL(Calculate_Exact_Fluxes_CUDA, dim1dGrid, dim1dBlock, 0, 0, Q_Ly, Q_Ry, F_y, nx, ny, nz, n_ghost, gama, 1, n_fields);
    hipLaunchKernelGGL(Calculate_Exact_Fluxes_CUDA, dim1dGrid, dim1dBlock, 0, 0, Q_Lz, Q_Rz, F_z, nx, ny, nz, n_ghost, gama, 2, n_fields);
    #endif //EXACT
    #ifdef ROE
    hipLaunchKernelGGL(Calculate_Roe_Fluxes_CUDA, dim1dGrid, dim1dBlock, 0, 0, Q_Lx, Q_Rx, F_x, nx, ny, nz, n_ghost, gama, 0, n_fields);
    hipLaunchKernelGGL(Calculate_Roe_Fluxes_CUDA, dim1dGrid, dim1dBlock, 0, 0, Q_Ly, Q_Ry, F_y, nx, ny, nz, n_ghost, gama, 1, n_fields);
    hipLaunchKernelGGL(Calculate_Roe_Fluxes_CUDA, dim1dGrid, dim1dBlock, 0, 0, Q_Lz, Q_Rz, F_z, nx, ny, nz, n_ghost, gama, 2, n_fields);
    #endif //ROE
    #ifdef HLLC
    hipLaunchKernelGGL(Calculate_HLLC_Fluxes_CUDA, dim1dGrid, dim1dBlock, 0, 0, Q_Lx, Q_Rx, F_x, nx, ny, nz, n_ghost, gama, 0, n_fields);
    hipLaunchKernelGGL(Calculate_HLLC_Fluxes_CUDA, dim1dGrid, dim1dBlock, 0, 0, Q_Ly, Q_Ry, F_y, nx, ny, nz, n_ghost, gama, 1, n_fields);
    hipLaunchKernelGGL(Calculate_HLLC_Fluxes_CUDA, dim1dGrid, dim1dBlock, 0, 0, Q_Lz, Q_Rz, F_z, nx, ny, nz, n_ghost, gama, 2, n_fields);
    #endif //HLLC
    #ifdef HLL
    hipLaunchKernelGGL(Calculate_HLL_Fluxes_CUDA, dim1dGrid, dim1dBlock, 0, 0, Q_Lx, Q_Rx, F_x, nx, ny, nz, n_ghost, gama, 0, n_fields);
    hipLaunchKernelGGL(Calculate_HLL_Fluxes_CUDA, dim1dGrid, dim1dBlock, 0, 0, Q_Ly, Q_Ry, F_y, nx, ny, nz, n_ghost, gama, 1, n_fields);
    hipLaunchKernelGGL(Calculate_HLL_Fluxes_CUDA, dim1dGrid, dim1dBlock, 0, 0, Q_Lz, Q_Rz, F_z, nx, ny, nz, n_ghost, gama, 2, n_fields);
    #endif //HLLC
    CudaCheckError();

    #ifdef DE
    // Compute the divergence of Vel before updating the conserved array, this solves synchronization issues when adding this term on Update_Conserved_Variables_3D
    hipLaunchKernelGGL(Partial_Update_Advected_Internal_Energy_3D, dim1dGrid, dim1dBlock, 0, 0,  dev_conserved, Q_Lx, Q_Rx, Q_Ly, Q_Ry, Q_Lz, Q_Rz, nx, ny, nz, n_ghost, dx, dy, dz,  dt, gama, n_fields );
    CudaCheckError();
    #endif


    // Step 6: Update the conserved variable array
    hipLaunchKernelGGL(Update_Conserved_Variables_3D, dim1dGrid, dim1dBlock, 0, 0, dev_conserved, Q_Lx, Q_Rx, Q_Ly, Q_Ry, Q_Lz, Q_Rz, F_x, F_y, F_z, nx, ny, nz, x_off, y_off, z_off, n_ghost, dx, dy, dz, xbound, ybound, zbound, dt, gama, n_fields, density_floor, dev_grav_potential);
    CudaCheckError();

    #ifdef DE
    hipLaunchKernelGGL(Select_Internal_Energy_3D, dim1dGrid, dim1dBlock, 0, 0, dev_conserved, nx, ny, nz, n_ghost, n_fields);
    hipLaunchKernelGGL(Sync_Energies_3D, dim1dGrid, dim1dBlock, 0, 0, dev_conserved, nx, ny, nz, n_ghost, gama, n_fields);
    CudaCheckError();
    #endif

    #ifdef TEMPERATURE_FLOOR
    hipLaunchKernelGGL(Apply_Temperature_Floor, dim1dGrid, dim1dBlock, 0, 0, dev_conserved, nx, ny, nz, n_ghost, n_fields, U_floor );
    CudaCheckError();
    #endif //TEMPERATURE_FLOOR
  return;

}


void Free_Memory_VL_3D(){

  // free the GPU memory
  cudaFree(dev_conserved);
  cudaFree(dev_conserved_half);
  cudaFree(Q_Lx);
  cudaFree(Q_Rx);
  cudaFree(Q_Ly);
  cudaFree(Q_Ry);
  cudaFree(Q_Lz);
  cudaFree(Q_Rz);
  cudaFree(F_x);
  cudaFree(F_y);
  cudaFree(F_z);

}

__global__ void Update_Conserved_Variables_3D_half(Real *dev_conserved, Real *dev_conserved_half, Real *dev_F_x, Real *dev_F_y,  Real *dev_F_z, int nx, int ny, int nz, int n_ghost, Real dx, Real dy, Real dz, Real dt, Real gamma, int n_fields, Real density_floor )
{
  Real dtodx = dt/dx;
  Real dtody = dt/dy;
  Real dtodz = dt/dz;
  int n_cells = nx*ny*nz;

  // get a global thread ID
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  int zid = tid / (nx*ny);
  int yid = (tid - zid*nx*ny) / nx;
  int xid = tid - zid*nx*ny - yid*nx;
  int id = xid + yid*nx + zid*nx*ny;

  int imo = xid-1 + yid*nx + zid*nx*ny;
  int jmo = xid + (yid-1)*nx + zid*nx*ny;
  int kmo = xid + yid*nx + (zid-1)*nx*ny;

  #ifdef DE
  Real d, d_inv, vx, vy, vz;
  Real vx_imo, vx_ipo, vy_jmo, vy_jpo, vz_kmo, vz_kpo, P, E, E_kin, GE;
  int ipo, jpo, kpo;
  #endif

  #ifdef DENSITY_FLOOR
  Real dens_0;
  #endif

  // threads corresponding to all cells except outer ring of ghost cells do the calculation
  if (xid > 0 && xid < nx-1 && yid > 0 && yid < ny-1 && zid > 0 && zid < nz-1)
  {
    #ifdef DE
    d  =  dev_conserved[            id];
    d_inv = 1.0 / d;
    vx =  dev_conserved[1*n_cells + id] * d_inv;
    vy =  dev_conserved[2*n_cells + id] * d_inv;
    vz =  dev_conserved[3*n_cells + id] * d_inv;
    //PRESSURE_DE
    E = dev_conserved[4*n_cells + id];
    GE = dev_conserved[(n_fields-1)*n_cells + id];
    E_kin = 0.5 * d * ( vx*vx + vy*vy + vz*vz );
    P = hydro_utilities::Get_Pressure_From_DE( E, E - E_kin, GE, gamma );
    P  = fmax(P, (Real) TINY_NUMBER);
    // P  = (dev_conserved[4*n_cells + id] - 0.5*d*(vx*vx + vy*vy + vz*vz)) * (gamma - 1.0);
    //if (d < 0.0 || d != d) printf("Negative density before half step update.\n");
    //if (P < 0.0) printf("%d Negative pressure before half step update.\n", id);
    ipo = xid+1 + yid*nx + zid*nx*ny;
    jpo = xid + (yid+1)*nx + zid*nx*ny;
    kpo = xid + yid*nx + (zid+1)*nx*ny;
    vx_imo = dev_conserved[1*n_cells + imo] / dev_conserved[imo];
    vx_ipo = dev_conserved[1*n_cells + ipo] / dev_conserved[ipo];
    vy_jmo = dev_conserved[2*n_cells + jmo] / dev_conserved[jmo];
    vy_jpo = dev_conserved[2*n_cells + jpo] / dev_conserved[jpo];
    vz_kmo = dev_conserved[3*n_cells + kmo] / dev_conserved[kmo];
    vz_kpo = dev_conserved[3*n_cells + kpo] / dev_conserved[kpo];
    #endif

    // update the conserved variable array
    dev_conserved_half[            id] = dev_conserved[            id]
                                       + dtodx * (dev_F_x[            imo] - dev_F_x[            id])
                                       + dtody * (dev_F_y[            jmo] - dev_F_y[            id])
                                       + dtodz * (dev_F_z[            kmo] - dev_F_z[            id]);
    dev_conserved_half[  n_cells + id] = dev_conserved[  n_cells + id]
                                       + dtodx * (dev_F_x[  n_cells + imo] - dev_F_x[  n_cells + id])
                                       + dtody * (dev_F_y[  n_cells + jmo] - dev_F_y[  n_cells + id])
                                       + dtodz * (dev_F_z[  n_cells + kmo] - dev_F_z[  n_cells + id]);
    dev_conserved_half[2*n_cells + id] = dev_conserved[2*n_cells + id]
                                       + dtodx * (dev_F_x[2*n_cells + imo] - dev_F_x[2*n_cells + id])
                                       + dtody * (dev_F_y[2*n_cells + jmo] - dev_F_y[2*n_cells + id])
                                       + dtodz * (dev_F_z[2*n_cells + kmo] - dev_F_z[2*n_cells + id]);
    dev_conserved_half[3*n_cells + id] = dev_conserved[3*n_cells + id]
                                       + dtodx * (dev_F_x[3*n_cells + imo] - dev_F_x[3*n_cells + id])
                                       + dtody * (dev_F_y[3*n_cells + jmo] - dev_F_y[3*n_cells + id])
                                       + dtodz * (dev_F_z[3*n_cells + kmo] - dev_F_z[3*n_cells + id]);
    dev_conserved_half[4*n_cells + id] = dev_conserved[4*n_cells + id]
                                       + dtodx * (dev_F_x[4*n_cells + imo] - dev_F_x[4*n_cells + id])
                                       + dtody * (dev_F_y[4*n_cells + jmo] - dev_F_y[4*n_cells + id])
                                       + dtodz * (dev_F_z[4*n_cells + kmo] - dev_F_z[4*n_cells + id]);
    #ifdef SCALAR
    for (int i=0; i<NSCALARS; i++) {
      dev_conserved_half[(5+i)*n_cells + id] = dev_conserved[(5+i)*n_cells + id]
                                         + dtodx * (dev_F_x[(5+i)*n_cells + imo] - dev_F_x[(5+i)*n_cells + id])
                                         + dtody * (dev_F_y[(5+i)*n_cells + jmo] - dev_F_y[(5+i)*n_cells + id])
                                         + dtodz * (dev_F_z[(5+i)*n_cells + kmo] - dev_F_z[(5+i)*n_cells + id]);
    }
    #endif
    #ifdef DE
    dev_conserved_half[(n_fields-1)*n_cells + id] = dev_conserved[(n_fields-1)*n_cells + id]
                                       + dtodx * (dev_F_x[(n_fields-1)*n_cells + imo] - dev_F_x[(n_fields-1)*n_cells + id])
                                       + dtody * (dev_F_y[(n_fields-1)*n_cells + jmo] - dev_F_y[(n_fields-1)*n_cells + id])
                                       + dtodz * (dev_F_z[(n_fields-1)*n_cells + kmo] - dev_F_z[(n_fields-1)*n_cells + id])
                                       + 0.5*P*(dtodx*(vx_imo-vx_ipo) + dtody*(vy_jmo-vy_jpo) + dtodz*(vz_kmo-vz_kpo));
    #endif

    #ifdef DENSITY_FLOOR
    if ( dev_conserved_half[            id] < density_floor ){
      dens_0 = dev_conserved_half[            id];
      printf("###Thread density change  %f -> %f \n", dens_0, density_floor );
      dev_conserved_half[            id] = density_floor;
      // Scale the conserved values to the new density
      dev_conserved_half[1*n_cells + id] *= (density_floor / dens_0);
      dev_conserved_half[2*n_cells + id] *= (density_floor / dens_0);
      dev_conserved_half[3*n_cells + id] *= (density_floor / dens_0);
      dev_conserved_half[4*n_cells + id] *= (density_floor / dens_0);
      #ifdef DE
      dev_conserved_half[(n_fields-1)*n_cells + id] *= (density_floor / dens_0);
      #endif
    }
    #endif
    //if (dev_conserved_half[id] < 0.0 || dev_conserved_half[id] != dev_conserved_half[id] || dev_conserved_half[4*n_cells+id] < 0.0 || dev_conserved_half[4*n_cells+id] != dev_conserved_half[4*n_cells+id]) {
      //printf("%3d %3d %3d Thread crashed in half step update. d: %e E: %e\n", xid, yid, zid, dev_conserved_half[id], dev_conserved_half[4*n_cells+id]);
    //}

  }

}




#endif //VL
#endif //CUDA
