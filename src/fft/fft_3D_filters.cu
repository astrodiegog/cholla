
#include <hip/hip_runtime.h>
#ifdef PARIS


#include "fft_3D.h"
#include "../utils/gpu.hpp"
#include "../io/io.h"
#include <cassert>
#include <cfloat>
#include <climits>


__host__ __device__ static inline double sqr(const double x) { return x*x; }

__device__ Real linear_interpolation( Real x, Real *x_vals, Real *y_vals, int N ){
  if ( x <= x_vals[0] ){
    printf(" x: %f  outside of interplation range.\n", x );
    return y_vals[0];
  }
  if ( x >= x_vals[N-1] ){
    printf(" x: %f  outside of interplation range.\n", x );
    return y_vals[N-1];
  }
  int indx = 0;
  while( x_vals[indx] < x ) indx +=1;
  // printf( "%d \n", indx );
  Real xl, xr, yl, yr;
  xl = x_vals[indx-1];
  xr = x_vals[indx];
  yl = y_vals[indx-1];
  yr = y_vals[indx];  
  if ( x < xl || x > xr ) printf(" ##################### Interpolation error:   x: %e  xl: %e  xr: %e   indx: %d\n", x, xl, xr, indx );
  return  yl + ( x - xl ) / ( xr - xl ) * ( yr - yl );
}




void FFT_3D::Filter_rescale_by_k_k2( double *input, double *output, bool in_device, int direction, double D ) const
{
  // Local copies of members for lambda capture
  const int ni = ni_, nj = nj_, nk = nk_;
  const double ddi = ddi_, ddj = ddj_, ddk = ddk_;
  const size_t bytes = minBytes_;

  if ( in_device ){
    CHECK( cudaMemcpy( db_, input, inputBytes_, cudaMemcpyDeviceToDevice));
  } else {
    CHECK( cudaMemcpy( db_, input, inputBytes_, cudaMemcpyHostToDevice));
  } 

  // Provide FFT filter with a lambda that multiplies by k / k^2 / D
  henry_->filter(bytes, db_, da_,
    [=] __device__ (const int i, const int j, const int k, const cufftDoubleComplex b) {
      if (i || j || k) {
      // const double kx = double(min(i,ni-i))*ddi;
      // const double ky = double(min(j,nj-j))*ddj;
      // const double kz = double(k)*ddk;
        int id_i = i < ni/2 ? i : i - ni;
        int id_j = j < nj/2 ? j : j - nj;
        int id_k = k < nk/2 ? k : k - nk;
        double kx = id_i * ddi;
        double ky = id_j * ddj;
        double kz = id_k * ddk;  
        double k2 = kx*kx + ky*ky + kz*kz ;
        if ( k2 == 0 ) k2 = 1.0;
        double factor;
        if      (direction == 0) factor = kx / k2 / D;
        else if (direction == 1) factor = ky / k2 / D;
        else if (direction == 2) factor = kz / k2 / D;
        else printf("Wrong direction %d\n", direction ); 
         // multiply b by 1j*factor ( Imaginary Number)
        return cufftDoubleComplex{-factor*b.y,factor*b.x};
      } else {
        return cufftDoubleComplex{0.0,0.0};
      }
    });
    
    if ( in_device ){
      CHECK( cudaMemcpy( output, da_, outputBytes_, cudaMemcpyDeviceToDevice));
    } else {
      CHECK( cudaMemcpy( output, da_, outputBytes_, cudaMemcpyDeviceToHost));
    } 
}

void FFT_3D::Filter_rescale_by_power_spectrum( double *input, double *output, bool in_device, int size, double *dev_k, double *dev_pk ) const
{
  // Local copies of members for lambda capture
  const int ni = ni_, nj = nj_, nk = nk_;
  const double ddi = ddi_, ddj = ddj_, ddk = ddk_;
  const size_t bytes = minBytes_;
  
  if ( in_device ){
    CHECK( cudaMemcpy( db_, input, inputBytes_, cudaMemcpyDeviceToDevice));
  } else {
    CHECK( cudaMemcpy( db_, input, inputBytes_, cudaMemcpyHostToDevice));
  } 
  
  // Provide FFT filter with a lambda that multiplies by P(k)
  henry_->filter(bytes, db_, da_,
    [=] __device__ (const int i, const int j, const int k, const cufftDoubleComplex b) {
      if (i || j || k) {
        // const double kx = double(min(i,ni-i))*ddi;
        // const double ky = double(min(j,nj-j))*ddj;
        // const double kz = double(k)*ddk;
        int id_i = i < ni/2 ? i : i - ni;
        int id_j = j < nj/2 ? j : j - nj;
        int id_k = k < nk/2 ? k : k - nk;
        double kx = id_i * ddi;
        double ky = id_j * ddj;
        double kz = id_k * ddk;  
        const double k_mag = sqrt( kx*kx + ky*ky + kz*kz );
        double pk = linear_interpolation( k_mag, dev_k, dev_pk, size );
        if ( i==1 && j==1 && k==1 ) printf("###### kx: %e  ky: %e  kz: %e  k_mag: %e  pk: %e \n", kx, ky, kz, k_mag, pk );  
        pk = sqrt(pk);
        return cufftDoubleComplex{pk*b.x,pk*b.y};
      } else {
        return cufftDoubleComplex{0.0,0.0};
      }
    });
    
    if ( in_device ){
      CHECK( cudaMemcpy( output, da_, outputBytes_, cudaMemcpyDeviceToDevice));
    } else {
      CHECK( cudaMemcpy( output, da_, outputBytes_, cudaMemcpyDeviceToHost));
    } 
    
}

void FFT_3D::Filter_inv_k2( double *const input, double *const output, bool in_device ) const
{
  // Local copies of members for lambda capture
  const int ni = ni_, nj = nj_;
  const double ddi = ddi_, ddj = ddj_, ddk = ddk_;
  const size_t bytes = minBytes_;
  
  if ( in_device ){
    CHECK( cudaMemcpy( db_, input, inputBytes_, cudaMemcpyDeviceToDevice));
  } else {
    CHECK( cudaMemcpy( db_, input, inputBytes_, cudaMemcpyHostToDevice));
  } 
    
  // Provide FFT filter with a lambda that does 1/k^2 solve in frequency space
  henry_->filter(bytes, db_, da_,
    [=] __device__ (const int i, const int j, const int k, const cufftDoubleComplex b) {
      if (i || j || k) {
        const double i2 = sqr(double(min(i,ni-i))*ddi);
        const double j2 = sqr(double(min(j,nj-j))*ddj);
        const double k2 = sqr(double(k)*ddk);
        const double d = -1.0/(i2+j2+k2);
        return cufftDoubleComplex{d*b.x,d*b.y};
      } else {
        return cufftDoubleComplex{0.0,0.0};
      }
    });
    
  if ( in_device ){
    CHECK( cudaMemcpy( output, da_, outputBytes_, cudaMemcpyDeviceToDevice));
  } else {
    CHECK( cudaMemcpy( output, da_, outputBytes_, cudaMemcpyDeviceToHost));
  } 
}



#endif

